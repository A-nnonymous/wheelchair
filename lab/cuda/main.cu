#include <random>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdexcept>
#include <vector>

// declarations
extern "C" __global__ void cinn_kernel(const float* src, const float* src_1, const float* src_10, float* dst);
extern "C" __global__ void simplified_kernel(const float* src, const float* src_1, const float* src_10, float* dst);

// CUDA call wraper
#define CUDA_CALL(call) {                                    \
    const hipError_t error = call;                           \
    if (error != hipSuccess) {                               \
        std::cerr << "Error: " << hipGetErrorString(error);  \
        throw std::runtime_error("CUDA API call failed");     \
    }                                                         \
}

int main() {
    try {
        // vector item size
        constexpr size_t srcSize = 1536 * 256 * 4;
        constexpr size_t dstSize = 1536*256 * 4;

        constexpr dim3 blocks(1536);
        constexpr dim3 threads(256);

        // DRAM buffer initialize
        std::vector<float> h_src(srcSize);
        std::vector<float> h_src_1(srcSize);
        std::vector<float> h_src_10(srcSize);
        std::vector<float> h_dst(dstSize);


        // random initialize
        std::uniform_real_distribution<float> dist(-__FLT_MAX__, __FLT_MAX__);
        std::mt19937_64 rng(std::random_device{}());
        #pragma loop_unroll(4)
        for(size_t i = 0; i < srcSize; ++i) {
          h_src[i] = dist(rng);
          h_src_1[i] = dist(rng);
          h_src_10[i] = dist(rng);
          h_dst[i] = dist(rng);
        }

        // device memory allocate
        float* d_src;
        float* d_src_1;
        float* d_src_10;
        float* d_dst;
        CUDA_CALL(hipMalloc(&d_src, srcSize * sizeof(float)));
        CUDA_CALL(hipMalloc(&d_src_1, srcSize * sizeof(float)));
        CUDA_CALL(hipMalloc(&d_src_10, srcSize * sizeof(float)));
        CUDA_CALL(hipMalloc(&d_dst, dstSize * sizeof(float)));

        // data xfer from host to device
        CUDA_CALL(hipMemcpy(d_src, h_src.data(), srcSize * sizeof(float), hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(d_src_1, h_src_1.data(), srcSize * sizeof(float), hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(d_src_10, h_src_10.data(), srcSize * sizeof(float), hipMemcpyHostToDevice));

        // summon cuda kernel
        //cinn_kernel<<<blocks, threads>>>(d_src, d_src_1, d_src_10, d_dst);
        simplified_kernel<<<blocks, threads>>>(d_src, d_src_1, d_src_10, d_dst);

        //  data xfer from device to host
        CUDA_CALL(hipMemcpy(h_dst.data(), d_dst, dstSize * sizeof(float), hipMemcpyDeviceToHost));

        // device mem free
        CUDA_CALL(hipFree(d_src));
        CUDA_CALL(hipFree(d_src_1));
        CUDA_CALL(hipFree(d_src_10));
        CUDA_CALL(hipFree(d_dst));


    } catch (const std::exception& e) {
        std::cerr << "Exception caught: " << e.what() << std::endl;
        return -1;
    }

    return 0;
}

extern "C" {

__global__
void __launch_bounds__(256) cinn_kernel(const float* __restrict__ src, const float* __restrict__ src_1, const float* __restrict__ src_10, float* __restrict__ dst)
{
  if (((int)blockIdx.x < 1536)) {
    for (int32_t i_j_k_a_fused_0 = 0; i_j_k_a_fused_0 < 4; i_j_k_a_fused_0 += 1) {
      if (((int)threadIdx.x < 256)) {
        float local_var = src_10[(((int)threadIdx.x & 127) + (((((((int)threadIdx.x / 128ll) + ((8ll * (int)blockIdx.x) + (2ll * i_j_k_a_fused_0))) / 128ll) / 96ll) * 1572864ll) + ((16384ll * (((((int)threadIdx.x / 128ll) + ((8ll * (int)blockIdx.x) + (2ll * i_j_k_a_fused_0))) / 128ll) % 96ll)) + (128ll * ((((int)threadIdx.x / 128ll) + ((8ll * (int)blockIdx.x) + (2ll * i_j_k_a_fused_0))) & 127)))))];
        float local_var_0 = src_1[(((((int)threadIdx.x / 128ll) + ((8ll * (int)blockIdx.x) + (2ll * i_j_k_a_fused_0))) / 128ll) % 96ll)];
        float local_var_1 = src[(((((int)threadIdx.x / 128ll) + ((8 * (int)blockIdx.x) + (2 * i_j_k_a_fused_0))) / 128ll) % 96ll)];
        dst[(((int)threadIdx.x & 127) + (((((((int)threadIdx.x / 128ll) + ((8ll * (int)blockIdx.x) + (2ll * i_j_k_a_fused_0))) / 128ll) / 96ll) * 1572864ll) + ((16384ll * (((((int)threadIdx.x / 128ll) + ((8ll * (int)blockIdx.x) + (2ll * i_j_k_a_fused_0))) / 128ll) % 96ll)) + (128ll * ((((int)threadIdx.x / 128ll) + ((8ll * (int)blockIdx.x) + (2ll * i_j_k_a_fused_0))) & 127)))))] = (local_var * (1.00000000f / (1.00000000f + exp(((-1.00000000f * (local_var_0 + local_var_1)) + 0.00000000f)))));
      };
    };
  };
}

__global__
void __launch_bounds__(256) simplified_kernel(const float* __restrict__ src, const float* __restrict__ src_1, const float* __restrict__ src_10, float* __restrict__ dst) {
  if (((int)blockIdx.x < 1536)) {
    for (int32_t i_j_k_a_fused_0 = 0; i_j_k_a_fused_0 < 4; i_j_k_a_fused_0 += 1) {
      if (((int)threadIdx.x < 256)) {
        float local_var = src_10[(((int)threadIdx.x & 127) + (((((((int)threadIdx.x / 128ll) + ((8ll * (int)blockIdx.x) + (2ll * i_j_k_a_fused_0))) / 128ll) / 96ll) * 1572864ll) + ((16384ll * (((((int)threadIdx.x / 128ll) + ((8ll * (int)blockIdx.x) + (2ll * i_j_k_a_fused_0))) / 128ll) % 96ll)) + (128ll * ((((int)threadIdx.x / 128ll) + ((8ll * (int)blockIdx.x) + (2ll * i_j_k_a_fused_0))) & 127)))))];
        float local_var_0 = src_1[(((((int)threadIdx.x / 128ll) + ((8ll * (int)blockIdx.x) + (2ll * i_j_k_a_fused_0))) / 128ll) % 96ll)];
        float local_var_1 = src[(((((int)threadIdx.x / 128ll) + ((8 * (int)blockIdx.x) + (2 * i_j_k_a_fused_0))) / 128ll) % 96ll)];
        dst[(((int)threadIdx.x & 127) + (((((((int)threadIdx.x / 128ll) + ((8ll * (int)blockIdx.x) + (2ll * i_j_k_a_fused_0))) / 128ll) / 96ll) * 1572864ll) + ((16384ll * (((((int)threadIdx.x / 128ll) + ((8ll * (int)blockIdx.x) + (2ll * i_j_k_a_fused_0))) / 128ll) % 96ll)) + (128ll * ((((int)threadIdx.x / 128ll) + ((8ll * (int)blockIdx.x) + (2ll * i_j_k_a_fused_0))) & 127)))))] = (local_var * (1.00000000f / (1.00000000f + exp(((-1.00000000f * (local_var_0 + local_var_1)) + 0.00000000f)))));
      };
    }
  }
}
}
